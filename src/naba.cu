
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add( int *a, int *b, int *c) {
	*c = *a + *b;
}

__global__ void array_add (int *a, int *b, int *c, int sz) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if ( idx < sz)
		c[idx] = a[idx] + b[idx];

}
int main(void) {
	int a, b, c;
	int *dev_a, *dev_b, *dev_c;
	int size = sizeof(int);
	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);

	a = 2;
	b = 7;
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
	add<<< 1, 1 >>>(dev_a, dev_b, dev_c);

	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", a, b, c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	size = sizeof(int) * 4096;

	int *x = (int *)malloc(size);
	int *y = (int *)malloc(size);
	int *z = (int *)malloc(size);
	
	memset(x, 1, size);
	memset(y, 2, size);

	hipMalloc((void **)&dev_a, size);
        hipMalloc((void **)&dev_b, size);
        hipMalloc((void **)&dev_c, size);

	x[0] = 5;
	y[0] = 5;

	x[1024] = 2;
	y[1024] = 2;

	x[2047] = 1;
	y[2047] = 1;
	hipMemcpy(dev_a, x, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, y, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 1024;
	int blocksPerGrid =
            (4096 + threadsPerBlock - 1) / threadsPerBlock;

	array_add<<< blocksPerGrid, threadsPerBlock >>>(dev_a, dev_b, dev_c, 4096);

	hipMemcpy(z, dev_c, size, hipMemcpyDeviceToHost);
	for (int j = 0; j < 4096; j++) {
		printf("%d ", z[j]);
	}
	printf("\n");

	free(x);
	free(y);
	free(z);

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);

}
