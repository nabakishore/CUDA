#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add( int *a, int *b, int *c) {
	*c = *a + *b;
}

__global__ void array_add (int *a, int *b, int *c, int sz) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if ( idx < sz)
		c[idx] = a[idx] + b[idx];

}
int main(void) {
	int a, b, c;
	int *dev_a, *dev_b, *dev_c;
	int size = sizeof(int);
	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);

	a = 2;
	b = 7;
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
	add<<< 1, 1 >>>(dev_a, dev_b, dev_c);

	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

	printf("%d + %d = %d\n", a, b, c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	size = sizeof(int) * 2048;

	int *x = malloc(size);
	int *y = malloc(size);

	memset(x, 1, 2048);
	memset(y, 2, 2048);

	hipMalloc((void **)&dev_a, size);
        hipMalloc((void **)&dev_b, size);
        hipMalloc((void **)&dev_c, size);

	hipMemcpy(dev_a, x, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_b, y, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 1024;
	int blocksPerGrid =
            (2048 + threadsPerBlock - 1) / threadsPerBlock;


}
